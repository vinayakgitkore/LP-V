
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *result, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        result[tid] = a[tid] + b[tid];
    }
}

int main() {
    int *a, *b, *c;
    int *a_dev, *b_dev, *c_dev;
    int n = 1 << 24; // Example large number

    a = new int[n];
    b = new int[n];
    c = new int[n];
    int size = n * sizeof(int);

    hipMalloc(&a_dev, size);
    hipMalloc(&b_dev, size);
    hipMalloc(&c_dev, size);

    // Initialize arrays with values
    for (int i = 0; i < n; i++) {
        a[i] = 1;
        b[i] = 2;
    }

    // Copy data from host to device
    hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    int threads = 1024;
    int blocks = (n + threads - 1) / threads;

    // Execute kernel on GPU
    vectorAdd<<<blocks, threads>>>(a_dev, b_dev, c_dev, n);

    // Copy result back to host
    hipMemcpy(c, c_dev, size, hipMemcpyDeviceToHost);

    // Verify the result
    bool success = true;
    for (int i = 0; i < n; i++) {
        if (c[i] != a[i] + b[i]) {
            success = false;
            std::cout << "Error at position " << i << std::endl;
            break;
        }
    }

    if (success) {
        std::cout << "Vector addition successful!" << std::endl;
    }

    // Free memory
    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    delete[] a;
    delete[] b;
    delete[] c;

    return 0;
}

// IF LOCALLY :
// Command to compile -> nvcc -o vectorAddition vectorAddition.cu
// Command to execute -> ./vectorAddition

// IF IN COLAB :
// Command to first install nvcc_plugin -> !pip install git+https://github.com/afnan47/cuda.git
// Then -> %load_ext nvcc_plugin
// Then write %%cu at the top and then copy the above entire code and run it.